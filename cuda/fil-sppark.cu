#include <hip/hip_runtime.h>

// Imports from sppark. The order matters.
#include <ff/bls12-381.hpp>
#include <ntt/ntt.cuh>
#include <ec/jacobian_t.hpp>
#include <ec/xyzz_t.hpp>
// Those definitions are needed by `msm/pippenger.cuh`.
typedef xyzz_t<fp_t> bucket_t;
typedef bucket_t::affine_t affine_t;
typedef fr_t scalar_t;
#include <msm/pippenger.cuh>

#ifndef __CUDA_ARCH__
extern "C" RustError mult_pippenger(jacobian_t<fp_t>* out,
                                    const affine_t points[],
                                    size_t npoints,
                                    const scalar_t scalars[]) {
   return mult_pippenger<bucket_t>(out, points, npoints, scalars, false);
}

//extern "C" RustError compute_ntt(fr_t* inout, uint32_t lg_domain_size,
//                     NTT::InputOutputOrder ntt_order,
//                     NTT::Direction ntt_direction,
//                     NTT::Type ntt_type)
extern "C" RustError compute_ntt(fr_t* inout, uint32_t lg_domain_size) {
   // As for MSM, use the first available device for now.
   auto& gpu = select_gpu(0);
   return NTT::Base(gpu, inout, lg_domain_size, NTT::InputOutputOrder::NN,
                    NTT::Direction::forward, NTT::Type::standard);
}
#endif
